#include "hip/hip_runtime.h"
__global__ void snonlin_gpu(int nx,int ny,cal_type* awk_array,cal_type cong,cal_type* fconst0_array
        ,cal_type* se_array,cal_type* dse_array,cal_type al13,cal_type al23,cal_type al11,cal_type al21,cal_type al31
        ,int k_length,int j_length,int klp1,int pnts_num,int pnts_num_calc)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = iy*nx + ix + 1;
    int pnt_id = 1;
    for(pnt_id = idx;pnt_id <= pnts_num_calc; pnt_id+=nx*ny)
    {
        cal_type depth=d_array[pnt_id-1];
        cal_type awk_value=awk_array[pnt_id-1];
        cal_type xx = 0.75*depth*awk_value;
        xx = xx>0.5?xx:0.5;
        cal_type enh = 1+(5.5/xx)*(1-0.833*xx)*exp(-1.25*xx);
        int kh=0;
        int k_iter=1,mr_iter=1,j_iter=1;
        for(k_iter=1;k_iter<=k_length;k_iter++)
        {
            cal_type wp11=wp_array[0*2*k_length+0*k_length+k_iter-1];
            cal_type wp12=wp_array[1*2*k_length+0*k_length+k_iter-1];
            cal_type wp21=wp_array[0*2*k_length+1*k_length+k_iter-1];
            cal_type wp22=wp_array[1*2*k_length+1*k_length+k_iter-1];
            cal_type wm11=wm_array[0*2*k_length+0*k_length+k_iter-1];
            cal_type wm12=wm_array[1*2*k_length+0*k_length+k_iter-1];
            cal_type wm21=wm_array[0*2*k_length+1*k_length+k_iter-1];
            cal_type wm22=wm_array[1*2*k_length+1*k_length+k_iter-1];
            int ip = ikp_array[k_iter-1];
            int ip1 = ikp1_array[k_iter-1];
            int im = ikm_array[k_iter-1];
            int im1 = ikm1_array[k_iter-1];
            int kp = ip;
            int kp1 = ip1;
            int kp2 = ip;
            int kp3 = ip1;
            cal_type ffacp = 1.0;
            cal_type ffacp1 = 1.0;
            cal_type cwks17 = cong * wks17_array[k_iter-1];
            if(kp>=k_length)
            {
                kh+=1;
                kp2=k_length+1;
                if(kp==k_length)
                {
                    kp2=k_length;
                }
                kp=k_length;
                kp1=k_length;
                kp3=k_length+1;
                ffacp = wkh_array[kh-1];
                ffacp1 = wkh_array[kh];
            }
            for(mr_iter=1;mr_iter<=2;mr_iter++)
            {
                for(j_iter=1;j_iter<=j_length;j_iter++)
                {
                    int j11 = jp1_array[(j_iter-1)*2+mr_iter-1];
                    int j12 = jp2_array[(j_iter-1)*2+mr_iter-1];
                    int j21 = jm1_array[(j_iter-1)*2+mr_iter-1];
                    int j22 = jm2_array[(j_iter-1)*2+mr_iter-1];
                    cal_type e_value = e_array[(pnt_id)*j_length*k_length+(j_iter-1)*k_length+(k_iter-1)];
                    if(e_value < 1e-20)
                        continue;
                    cal_type ea1 = e_array[(pnt_id)*j_length*k_length+(j11-1)*k_length+kp-1];
                    cal_type ea2 = e_array[(pnt_id)*j_length*k_length+(j12-1)*k_length+kp-1];
                    cal_type ea3 = e_array[(pnt_id)*j_length*k_length+(j11-1)*k_length+kp1-1];
                    cal_type ea4 = e_array[(pnt_id)*j_length*k_length+(j12-1)*k_length+kp1-1];
                    cal_type ea5 = e_array[(pnt_id)*j_length*k_length+(j21-1)*k_length+im-1];
                    cal_type ea6 = e_array[(pnt_id)*j_length*k_length+(j22-1)*k_length+im-1];
                    cal_type ea7 = e_array[(pnt_id)*j_length*k_length+(j21-1)*k_length+im1-1];
                    cal_type ea8 = e_array[(pnt_id)*j_length*k_length+(j22-1)*k_length+im1-1];

                    cal_type up = (wp11*ea1+wp12*ea2)*ffacp;
                    cal_type up1 = (wp21*ea3+wp22*ea4)*ffacp1;
                    cal_type um = wm11*ea5+wm12*ea6;
                    cal_type um1 = wm21*ea7+wm22*ea8;
                    cal_type sap = up+up1;
                    cal_type sam = um+um1;
                    cal_type e_square = pow(e_value,2);
                    cal_type zua=2.0*e_value/al31;
                    cal_type ead1=sap/al11+sam/al21;
                    cal_type ead2=-2.0*sap*sam/al31;
                    cal_type fcen=fconst0_array[(pnt_id-1)*k_length+k_iter-1]*enh;
                    cal_type ad=cwks17*(e_square*ead1+ead2*e_value)*fcen;
                    cal_type adp=ad/al13;
                    cal_type adm=ad/al23;
                    cal_type delad=cwks17*(e_value*2*ead1+ead2)*fcen;
                    cal_type deladp=cwks17*(e_square/al11-zua*sam)*fcen/al13;
                    cal_type deladm=cwks17*(e_square/al21-zua*sap)*fcen/al23;
                    se_array[(pnt_id-1)*klp1*j_length+(j_iter-1)*klp1+k_iter-1] -= 2*ad;
                    se_array[(pnt_id-1)*klp1*j_length+(j11-1)*klp1+kp2-1] += adp * wp11;
                    se_array[(pnt_id-1)*klp1*j_length+(j12-1)*klp1+kp2-1] += adp * wp12;
                    se_array[(pnt_id-1)*klp1*j_length+(j11-1)*klp1+kp3-1] += adp * wp21;
                    se_array[(pnt_id-1)*klp1*j_length+(j12-1)*klp1+kp3-1] += adp * wp22;
                    se_array[(pnt_id-1)*klp1*j_length+(j21-1)*klp1+im-1] += adm * wm11;
                    se_array[(pnt_id-1)*klp1*j_length+(j22-1)*klp1+im-1] += adm * wm12;
                    se_array[(pnt_id-1)*klp1*j_length+(j21-1)*klp1+im1-1] += adm * wm21;
                    se_array[(pnt_id-1)*klp1*j_length+(j22-1)*klp1+im1-1] += adm * wm22;
                    dse_array[(pnt_id-1)*klp1*j_length+(j_iter-1)*klp1+k_iter-1] -= 2*delad;
                    dse_array[(pnt_id-1)*klp1*j_length+(j11-1)*klp1+kp2-1] += deladp * pow(wp11,2);
                    dse_array[(pnt_id-1)*klp1*j_length+(j12-1)*klp1+kp2-1] += deladp * pow(wp12,2);
                    dse_array[(pnt_id-1)*klp1*j_length+(j11-1)*klp1+kp3-1] += deladp * pow(wp21,2);
                    dse_array[(pnt_id-1)*klp1*j_length+(j12-1)*klp1+kp3-1] += deladp * pow(wp22,2);
                    dse_array[(pnt_id-1)*klp1*j_length+(j21-1)*klp1+im-1] += deladm * pow(wm11,2);
                    dse_array[(pnt_id-1)*klp1*j_length+(j22-1)*klp1+im-1] += deladm * pow(wm12,2);
                    dse_array[(pnt_id-1)*klp1*j_length+(j21-1)*klp1+im1-1] += deladm * pow(wm21,2);
                    dse_array[(pnt_id-1)*klp1*j_length+(j22-1)*klp1+im1-1] += deladm * pow(wm22,2);
                }
            }
        }
    }
}